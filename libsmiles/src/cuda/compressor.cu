#include "hip/hip_runtime.h"
#include "compression_dictionary.hpp"
#include "cpu/compressor.hpp"
#include "cuda/compressor.cuh"
#include "cuda/nvidia_helper.cuh"
#include "utils.hpp"

#include <cassert>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <string_view>

#define WARP_SIZE  32
#define BLOCK_SIZE WARP_SIZE
#define GRID_SIZE  512

#define MAX_SMILES_LEN   512
#define SMILES_PER_BLOCK 128

#define SMILES_PER_DEVICE GRID_SIZE* BLOCK_SIZE* SMILES_PER_BLOCK
#define CHAR_PER_DEVICE   SMILES_PER_DEVICE* MAX_SMILES_LEN

namespace smiles {
  namespace cuda {
    __device__ __constant__ node dictionary_tree_gpu[425];

    smiles_compressor::smiles_compressor() {
      smiles_host.reserve(CHAR_PER_DEVICE);
      CHECK_CUDA_KERNEL_ERRORS(hipMalloc(&smiles_dev, CHAR_PER_DEVICE * sizeof(smiles_type)));
      CHECK_CUDA_KERNEL_ERRORS(hipMalloc(&smiles_len_dev, SMILES_PER_DEVICE * sizeof(index_type)));
      CHECK_CUDA_KERNEL_ERRORS(
          hipMalloc(&match_matrix_dev,
                     MAX_SMILES_LEN * GRID_SIZE * LONGEST_PATTERN * sizeof(pattern_index_type)));
      CHECK_CUDA_KERNEL_ERRORS(
          hipMalloc(&dijkstra_matrix_dev,
                     MAX_SMILES_LEN * GRID_SIZE * LONGEST_PATTERN * sizeof(pattern_index_type)));
      CHECK_CUDA_KERNEL_ERRORS(hipMalloc(&smiles_output_dev, CHAR_PER_DEVICE * sizeof(smiles_type)));
      smiles_output_host.resize(CHAR_PER_DEVICE);
      CHECK_CUDA_KERNEL_ERRORS(hipMemcpyToSymbol(HIP_SYMBOL(dictionary_tree_gpu),
                                                  build_gpu_smiles_dictionary().data(),
                                                  sizeof(node) * 425,
                                                  0,
                                                  hipMemcpyHostToDevice));
    };

    smiles_compressor::~smiles_compressor() {
      if (smiles_dev != nullptr)
        CHECK_CUDA_KERNEL_ERRORS(hipFree(smiles_dev));
      if (smiles_len_dev != nullptr)
        CHECK_CUDA_KERNEL_ERRORS(hipFree(smiles_len_dev));
      if (match_matrix_dev != nullptr)
        CHECK_CUDA_KERNEL_ERRORS(hipFree(match_matrix_dev));
      if (dijkstra_matrix_dev != nullptr)
        CHECK_CUDA_KERNEL_ERRORS(hipFree(dijkstra_matrix_dev));
      if (smiles_output_dev != nullptr)
        CHECK_CUDA_KERNEL_ERRORS(hipFree(smiles_output_dev));
    }

    __global__ void compress_gpu(const smiles_compressor::smiles_type* __restrict__ smiles_in,
                                 smiles_compressor::smiles_type* __restrict__ smiles_out,
                                 const smiles_compressor::index_type* __restrict__ smiles_len,
                                 const int num_smiles,
                                 smiles_compressor::pattern_index_type* __restrict__ match_matrix,
                                 smiles_compressor::pattern_index_type* __restrict__ dijkstra_matrix,
                                 const int min_costs_stride) {
      const int threadId     = threadIdx.x;
      const int blockId      = blockIdx.x;
      const int stride_smile = gridDim.x;
      const int stride       = blockDim.x;
      const int offset       = MAX_SMILES_LEN * blockId;

      __shared__ smiles_compressor::smiles_type smiles_s[MAX_SMILES_LEN];

      const int* smiles_len_l                                  = smiles_len + blockId;
      const smiles_compressor::smiles_type* smiles_in_l        = smiles_in + offset;
      smiles_compressor::smiles_type* smiles_out_l             = smiles_out + offset;
      smiles_compressor::pattern_index_type* match_matrix_l    = match_matrix + offset;
      smiles_compressor::pattern_index_type* dijkstra_matrix_l = dijkstra_matrix + offset;
      for (int id = blockId; id < num_smiles; id += stride_smile,
               smiles_len_l += stride_smile,
               smiles_in_l += MAX_SMILES_LEN,
               smiles_out_l += MAX_SMILES_LEN) {
        const int smile_len = *smiles_len_l;
#pragma unroll 8
        for (int i = threadId; i < smile_len; i += stride) smiles_s[i] = smiles_in_l[i];
        __syncwarp();
#pragma unroll 8
        for (int i = 0; i < LONGEST_PATTERN; i++)
          for (int j = threadId; j < smile_len; j += stride) match_matrix_l[min_costs_stride * i + j] = 0;
        __syncwarp();
        // For each position in the input string

        for (int i = threadId; i < smile_len; i += stride) {
          const node* curr = dictionary_tree_gpu;
          int curr_id      = 0;
#pragma unroll 8
          for (int j = 0; j < LONGEST_PATTERN && curr && j < (smile_len - i); j++) {
            const int next_i = curr->neighbor[smiles_s[i + j] - NOT_PRINTABLE];
            if (next_i) {
              curr    = &dictionary_tree_gpu[next_i + curr_id];
              curr_id = next_i + curr_id;
              if (curr->pattern != -1)
              // TODO change this matrix to be column major
                match_matrix_l[i + j + 1 + min_costs_stride * j] = curr->pattern;
            } else {
              curr = nullptr;
            }
          }
        }
#pragma unroll 8
        for (int i = 0; i < LONGEST_PATTERN; i++)
          for (int j = threadId; j < smile_len; j += stride) {
            dijkstra_matrix_l[min_costs_stride * i + j] =
                std::numeric_limits<smiles_compressor::pattern_index_type>().max();
          }
        __syncwarp();
        if (threadId % stride == 0) {
          dijkstra_matrix_l[smile_len]                        = 0;
          dijkstra_matrix_l[min_costs_stride + smile_len]     = 0;
          dijkstra_matrix_l[min_costs_stride * 2 + smile_len] = 0;

          // Skip the first one which is trivial to select the smallest value
          for (int l = smile_len; l > 0; l--) {
            // Save the index of the prev first element of tot_cost into global memory
            // min_index[smile_end - l]        = cost_s[LONGEST_PATTERN];
            // min_index_output[smile_end - l] = cost_s[LONGEST_PATTERN * 2];
            smiles_compressor::pattern_index_type* costs_index_temp = match_matrix_l + l;
            smiles_compressor::cost_type best_costs                 = dijkstra_matrix_l[l] + 2;
            smiles_compressor::cost_type best_index                 = 0;

// Compute the best for the next one
#pragma unroll 8
            for (int t = 0; t < LONGEST_PATTERN; t++) {
              if (*costs_index_temp) {
                dijkstra_matrix_l[min_costs_stride * t + l - (t + 1)] = dijkstra_matrix_l[l] + 1;
              }
              costs_index_temp += min_costs_stride;
              if (best_costs > dijkstra_matrix_l[min_costs_stride * t + l - 1]) {
                best_index = t;
                best_costs = dijkstra_matrix_l[min_costs_stride * t + l - 1];
              }
            }
            dijkstra_matrix_l[l - 1] = dijkstra_matrix_l[min_costs_stride * best_index + l - 1];
            dijkstra_matrix_l[min_costs_stride + l - 1] = best_index;
            dijkstra_matrix_l[min_costs_stride * 2 + l - 1] =
                match_matrix_l[min_costs_stride * best_index + l + best_index];
          }
        }
        __syncwarp();
        // TODO you can parallelize and then make a reduction performed only by threadID 0
        if (threadId % stride == 0) {
          int o = 0;
          for (int l = 0; l < smile_len; l++ ) {
            if (!dijkstra_matrix_l[min_costs_stride * 2 + l] && !dijkstra_matrix_l[min_costs_stride + l]) {
              smiles_out_l[o] = '\\';
              o++;
              smiles_out_l[o] = smiles_s[l];
              o++;
            } else {
              smiles_out_l[o] =
                  static_cast<smiles_compressor::smiles_type>(dijkstra_matrix_l[min_costs_stride * 2 + l]);
              o++;
              l += dijkstra_matrix_l[min_costs_stride + l];
            }
          }
          smiles_out_l[o] = '\0';
          // printf("%s\n", &smiles_out[smile_start]);
        }
        __syncwarp();
      }
    }

    void smiles_compressor::compute_host(std::ofstream& out_s) {
      CHECK_CUDA_KERNEL_ERRORS(hipMemcpy(smiles_dev,
                                          smiles_host.data(),
                                          smiles_len.size() * MAX_SMILES_LEN * sizeof(smiles_type),
                                          hipMemcpyHostToDevice));
      CHECK_CUDA_KERNEL_ERRORS(hipMemcpy(smiles_len_dev,
                                          smiles_len.data(),
                                          smiles_len.size() * sizeof(int),
                                          hipMemcpyHostToDevice));

      const dim3 block_dimension{BLOCK_SIZE};
      const dim3 grid_dimension{GRID_SIZE};
      compress_gpu<<<grid_dimension, block_dimension>>>(smiles_dev,
                                                        smiles_output_dev,
                                                        smiles_len_dev,
                                                        smiles_len.size(),
                                                        match_matrix_dev,
                                                        dijkstra_matrix_dev,
                                                        MAX_SMILES_LEN * GRID_SIZE);
      CHECK_CUDA_ERRORS();
      hipDeviceSynchronize();

      // The copy back is SYNC
      CHECK_CUDA_KERNEL_ERRORS(hipMemcpy((void*) smiles_output_host.data(),
                                          smiles_output_dev,
                                          smiles_len.size() * MAX_SMILES_LEN * sizeof(smiles_type),
                                          hipMemcpyDeviceToHost));

      // Print output
      // -1 because we've also added the last one
      for (int i = 0; i < smiles_len.size(); i++) {
        out_s << &smiles_output_host.data()[i * MAX_SMILES_LEN] << std::endl;
      }
      // Clean up
      smiles_len.clear();
      smiles_host.clear();
      smiles_output_host.clear();

      return;
    }

    void smiles_compressor::clean_up(std::ofstream& out_s) {
      compute_host(out_s);
      return;
    }

    // we model the problem of SMILES compression as choosing the minimum path between the first character and the
    // last one. The cost of each path is the number of character that we need to produce in the output. We solve
    // this problem using Dijkstra and we use a support tree to perform pattern matching.
    void smiles_compressor::operator()(const std::string_view& plain_description, std::ofstream& out_s) {
      if (smiles_len.size() >= SMILES_PER_DEVICE) {
        return compute_host(out_s);
      }
      assert(plain_description.size() < MAX_SMILES_LEN);
      smiles_len.push_back(plain_description.size());
      smiles_host.append(plain_description);
      smiles_host.insert(smiles_host.end(), MAX_SMILES_LEN - plain_description.size(), '\0');
      return;
    }

    std::string_view smiles_decompressor::operator()(const std::string_view& compressed_description) {
      // decompressing a SMILES is really just a look up on the SMILES_DICTIONARY. We just need to pay attention
      // when the compressed SMILES has excaped something
      // NOTE: we need to start from a clean string
      // TODO
      return {};
    }
  } // namespace cuda
} // namespace smiles